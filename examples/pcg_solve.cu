#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "gpu_pcg.cuh"
#include "gpuassert.cuh"
#include "read_array.h"
#include <ctime>

#define tic      double tic_t = clock();
#define toc      std::cout << (clock() - tic_t)/CLOCKS_PER_SEC \
                           << " seconds" << std::endl;

int main() {

    const uint32_t state_size = STATE_SIZE;
    const uint32_t knot_points = KNOT_POINTS;
    const int Nnx2 = knot_points * state_size * state_size;
    const int Nnx = state_size * knot_points;

    double h_lambda[Nnx];
    for (int i = 0; i < Nnx; i++) {
        h_lambda[i] = 0;
    }
    double h_gamma[Nnx];
    double *h_S, *h_Pinv;
    std::string file_name_pre;

    if (PCG_TYPE) {
        // TRANS
        file_name_pre = "data/I_H_tilde_";
        h_S = new double[2 * Nnx2 + Nnx];
        h_Pinv = new double[2 * Nnx2 + Nnx];
        readArrayFromFile(2 * Nnx2, "data/Pob.txt", h_Pinv + Nnx);
        readArrayFromFile(Nnx, "data/Pdb.txt", h_Pinv);
        readArrayFromFile(2 * Nnx2, "data/Sob.txt", h_S + Nnx);
        readArrayFromFile(Nnx, "data/Sdb.txt", h_S);
        readArrayFromFile(Nnx, "data/gamma_tilde.txt", h_gamma);
    } else {
        // ORG
        file_name_pre = "data/I_H_";
        h_S = new double[3 * Nnx2];
        h_Pinv = new double[3 * Nnx2];
        readArrayFromFile(3 * Nnx2, "data/S.txt", h_S);
        readArrayFromFile(3 * Nnx2, "data/P.txt", h_Pinv);
        readArrayFromFile(Nnx, "data/gamma.txt", h_gamma);
    }


    struct pcg_config<double> config;
    config.pcg_org_trans = PCG_TYPE;
    config.pcg_poly_order = PRECOND_POLY_ORDER;

    if (PRECOND_POLY_ORDER == 1) {
        double h_I_H[3 * Nnx2];

        // information of alpha should match with MATLAB file
        int alpha_length = 9;
        double alpha_array[alpha_length];
        for (int i = 0; i < alpha_length; i++) {
            alpha_array[i] = 1 + i * 0.5;
        }

        for (int i = 0; i < alpha_length; i++) {
            double alpha = alpha_array[i];
            std::string file_name = file_name_pre + std::to_string(i + 1) + ".txt";
            const char *all = file_name.c_str();
            printf("reading from file %s\n", all);
            readArrayFromFile(3 * Nnx2, all, h_I_H);
            uint32_t res = solvePCG<double>(h_S,
                                           h_Pinv,
                                           h_I_H,
                                           h_gamma,
                                           h_lambda,
                                           state_size,
                                           knot_points,
                                           &config);
            double norm = 0;
            for (int i = 0; i < Nnx; i++) {
                norm += h_lambda[i] * h_lambda[i];
                h_lambda[i] = 0;
            }

            printf("summary of PCG %s\n", PCG_TYPE ? "TRANS" : "ORG");
            printf("type of preconditioner: %s\n", PRECOND_POLY_ORDER == 1 ? "p1s3" : "p0s3");
            printf("alpha = %f\n", alpha);
            printf("result: lambda norm = %f, pcg iter = %d\n\n", sqrt(norm), res);
        }

    } else if (PRECOND_POLY_ORDER == 0) {
        double *h_I_H = NULL;
        uint32_t res = solvePCG<double>(h_S,
                                       h_Pinv,
                                       h_I_H,
                                       h_gamma,
                                       h_lambda,
                                       state_size,
                                       knot_points,
                                       &config);
        double norm = 0;
        for (int i = 0; i < Nnx; i++) {
            norm += h_lambda[i] * h_lambda[i];
        }

        printf("summary of PCG ORG\n");
        printf("type of preconditioner: %s\n", PRECOND_POLY_ORDER == 1 ? "p1s3" : "p0s3");
        printf("result: lambda norm = %f, pcg iter = %d\n", sqrt(norm), res);
    }

    delete (h_S);
    delete (h_Pinv);

    return 0;
}

