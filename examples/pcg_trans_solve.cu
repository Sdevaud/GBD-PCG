#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "gpu_pcg.cuh"
#include "gpuassert.cuh"
#include "read_array.h"
#include <ctime>

#define tic      double tic_t = clock();
#define toc      std::cout << (clock() - tic_t)/CLOCKS_PER_SEC \
                           << " seconds" << std::endl;

int main(int argc, char *argv[]) {

    const uint32_t state_size = STATE_SIZE;
    const uint32_t knot_points = KNOT_POINTS;
    const int matrix_size = 2 * knot_points * state_size * state_size;
    const int vector_size = state_size * knot_points;

    float h_Pinvob[matrix_size];
    float h_Pinvdb[vector_size];
    float h_Sob[matrix_size];
    float h_Sdb[vector_size];

    readArrayFromFile(matrix_size, "data/Pob.txt", h_Pinvob);
    readArrayFromFile(vector_size, "data/Pdb.txt", h_Pinvdb);
    readArrayFromFile(matrix_size, "data/Sob.txt", h_Sob);
    readArrayFromFile(vector_size, "data/Sdb.txt", h_Sdb);

    float h_gamma[vector_size];
    readArrayFromFile(vector_size, "data/gamma_tilde.txt", h_gamma);
    float h_lambda[vector_size];
    for (int i = 0; i < vector_size; i++) {
        h_lambda[i] = 0;
    }

    struct pcg_config<float> config;
    config.pcg_poly_order = PRECOND_POLY_ORDER;
    if (PRECOND_POLY_ORDER == 1) {
        const int matrixH_size = 3 * knot_points * state_size * state_size;
        float h_H[matrixH_size];

        // information of alpha should match with MATLAB file
        int alpha_length = 9;
        float alpha_array[alpha_length];
        for (int i = 0; i < alpha_length; i++) {
            alpha_array[i] = 1 + i * 0.5;
        }

        for (int i = 0; i < alpha_length; i++) {
            float alpha = alpha_array[i];
            std::string file_name = "data/I_H_tilde_";
            file_name = file_name + std::to_string(i + 1) + ".txt";
            const char *all = file_name.c_str();
            printf("reading from file %s\n", all);
            readArrayFromFile(matrixH_size, all, h_H);
            uint32_t res = solvePCGTrans<float>(h_Sdb,
                                                h_Sob,
                                                h_Pinvdb,
                                                h_Pinvob,
                                                h_H,
                                                h_gamma,
                                                h_lambda,
                                                state_size,
                                                knot_points,
                                                &config);
            float norm = 0;
            for (int i = 0; i < vector_size; i++) {
                norm += h_lambda[i] * h_lambda[i];
                h_lambda[i] = 0;
            }
            printf("summary of PCG TRANS\n");
            printf("type of preconditioner: %s\n", PRECOND_POLY_ORDER == 1 ? "p1s3" : "p0s3");
            printf("alpha = %f\n", alpha);
            printf("result: lambda norm = %f, pcg iter = %d\n\n", sqrt(norm), res);
        }
    } else if (PRECOND_POLY_ORDER == 0) {
        float *h_H = NULL;
        uint32_t res = solvePCGTrans<float>(h_Sdb,
                                            h_Sob,
                                            h_Pinvdb,
                                            h_Pinvob,
                                            h_H,
                                            h_gamma,
                                            h_lambda,
                                            state_size,
                                            knot_points,
                                            &config);
        float norm = 0;
        for (int i = 0; i < vector_size; i++) {
            norm += h_lambda[i] * h_lambda[i];
        }
        printf("summary of PCG TRANS\n");
        printf("type of preconditioner: %s\n", PRECOND_POLY_ORDER == 1 ? "p1s3" : "p0s3");
        printf("result: lambda norm = %f, pcg iter = %d\n", sqrt(norm), res);
    }

    return 0;
}

